#include "hip/hip_runtime.h"


#include "cuda/CuPolicy.h"

CuPolicy::CuPolicy(paramfile &Param)
  {
    res.first = Param.find<int>("xres",800);
    res.second = Param.find<int>("yres",res.first);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    p_blockSize = deviceProp.maxThreadsPerBlock;
    m_gridSize = deviceProp.maxGridSize[0];
    gmsize = deviceProp.totalGlobalMem;

    size_t fbsize_def = (size_t) p_blockSize; //min(res.first/20, p_blockSize);
    fbsize_def *= m_gridSize*sizeof(cu_color);
    if ((8*fbsize_def) > gmsize) fbsize_def = gmsize/8;
    fbsize = Param.find<int>("fragment_buffer_size", fbsize_def);
    pix_blockSize = fbsize/(m_gridSize*sizeof(cu_color));
  }

pair<int,int> CuPolicy::GetResolution()
  {
    return res;
  }

size_t CuPolicy::GetFBufSize() // return dimension in terms of bytes
  {
     return fbsize; 
  }

size_t CuPolicy::GetIndexSize() // return dimension in terms of bytes
  {
     int npixels = m_gridSize*pix_blockSize; // (int) fbsize/sizeof(cu_color);
     size_t size = npixels*sizeof(int);
     return size; 
  }

size_t CuPolicy::GetGMemSize() // return dimension in terms of bytes
  { 
   // int MB = 1<<20;
   // int size = gmsize/MB;
    return gmsize; 
  }

int CuPolicy::GetMaxGridSize() 
  { 
    return m_gridSize; 
  }

int CuPolicy::GetMaxBlockSize()
  { 
    return pix_blockSize; 
  }

size_t CuPolicy::GetImageSize()
{
   // int MB = 1<<20;
    size_t size = (res.first)*(res.second)*sizeof(cu_color);
    return size;
}

void CuPolicy::GetDimsBlockGrid(int n, dim3 *dimGrid, dim3 *dimBlock)
  {
    *dimBlock = dim3(p_blockSize);
    int nBlock = (n + p_blockSize - 1)/p_blockSize;
    *dimGrid =dim3(nBlock); 
  }
