#include "hip/hip_runtime.h"
#ifndef SPLOTCH_KERNEL_H
#define SPLOTCH_KERNEL_H
/*
Try accelerating splotch with CUDA. July 2009.
Copyright things go here.
*/

#include "splotch_cuda.h"

//MACROs
#define Pi 3.14159265358979323846264338327950288
#define get_xy_from_sn(sn, xmin, ymin, ymax, x, y)\
        {int x1 =sn/(ymax-ymin); int y1 =sn-x1*(ymax-ymin);\
         x  =x1 +xmin; y  =y1 +ymin;}
#define get_sn_from_xy(x,y,maxy,miny, sn)\
    {sn =x*(maxy-miny) +y;}

/////////help functions///////////////////////////////////
__device__ float    my_asinh(float val)
  {
  return log(val+sqrt(1.+val*val));
  }

__device__ void my_normalize(float minv, float maxv, float &val)
  {
  if (minv!=maxv) val =  (val-minv)/(maxv-minv);
  }

__device__ void clamp (float minv, float maxv, float &val)
  {
  val = min(maxv, max(minv, val));
  }

//fetch a color from color table on device
__device__ cu_color get_color
  (int ptype, float val, cu_colormap_info info)
  {
  //copy things to local block memory
  __shared__ cu_color_map_entry *map;
  __shared__ int      mapSize;
  __shared__ int *ptype_points;
  __shared__ int ptypes;

  map =info.map;
  mapSize =info.mapSize;
  ptype_points =info.ptype_points;
  ptypes  =info.ptypes;

  cu_color        clr;
  clr.r =clr.g =clr.b =0.0;

  //first find the right entry for this ptype
  if (ptype>=ptypes)
    return clr; //invalid input
  int     start, end;
  start =ptype_points[ptype];
  if ( ptype == ptypes-1)//the last type
    end =mapSize-1;
  else
    end =ptype_points[ptype+1]-1;

  //search the section of this type to find the val
  int i=start;
  while ((val>map[i+1].val) && (i<end))
    ++i;

  float fract = (val-map[i].val)/(map[i+1].val-map[i].val);
  cu_color clr1=map[i].color, clr2=map[i+1].color;
  clr.r =clr1.r + fract*(clr2.r-clr1.r);
  clr.g =clr1.g + fract*(clr2.g-clr1.g);
  clr.b =clr1.b + fract*(clr2.b-clr1.b);

  return clr;
  }

__device__  float get_exp(float arg, cu_exptable_info d_exp_info)
  {
#if 0
  return exp(arg);
#else
  //fetch things to local
  __shared__  float   expfac;
  __shared__  float   *tab1, *tab2;
  __shared__  int     mask1, mask3, nbits;
  expfac  =d_exp_info.expfac;
  tab1    =d_exp_info.tab1;
  tab2    =d_exp_info.tab2;
  mask1   =d_exp_info.mask1;
  mask3   =d_exp_info.mask3;
  nbits   =d_exp_info.nbits;

  int iarg= (int)(arg*expfac);
  //  for final device code
  if (iarg&mask3)
    return (iarg<0) ? 1. : 0.;
  return tab1[iarg>>nbits]*tab2[iarg&mask1];
#endif
  }

__global__ void k_post_process(cu_color *pic, int n, cu_exptable_info exp_info)
  {
  //first get the index m of this thread
  int m=blockIdx.x *blockDim.x + threadIdx.x;
  if (m >=n)
    m =n;

  //each pic[m] should do the same calc, so sequence does not matter!
  pic[m].r =1.0 -get_exp( pic[m].r, exp_info);
  pic[m].g =1.0 -get_exp( pic[m].g, exp_info);
  pic[m].b =1.0 -get_exp( pic[m].b, exp_info);
  }

__global__ void k_combine
  (int minx, int miny, int maxx, int maxy, int xres, int yres,
  cu_particle_splotch *p, int pStart, int pEnd, cu_fragment_AeqE *fbuf, cu_color *pic)
  {
  int m =blockIdx.x *blockDim.x + threadIdx.x;
  int n =(maxx-minx)*(maxy-miny);
  if (m >=n)
    m =n;

  //get global coordinate point(x,y) of this thread
  int point_x, point_y;
  get_xy_from_sn(m, minx, miny, maxy, point_x, point_y);

  //go through all particles, for each particle p if point(x,y) is in its region
  //p(minx,miny, maxx,maxy) do the following.
  //find the sequencial number sn1 in p(minx,miny, maxx,maxy), the fragment we are looking
  //for in fragment buffer is fragBuf[ sn1+p.posInFBuf ]
  //grab the fragment f(deltaR,deltaG,deltaB)
  //find the sequencial number sn2 of point(x,y) in the output pic.
  //pic[sn2] += f
  int sn1, sn2, local_x, local_y, fpos;
  for (int i=pStart; i<=pEnd; i++)
    {
    if ( point_x >=p[i].minx && point_x<p[i].maxx &&
         point_y >=p[i].miny && point_y<p[i].maxy)
      {
      local_x =point_x -p[i].minx;
      local_y =point_y -p[i].miny;
      get_sn_from_xy(local_x, local_y, p[i].maxy, p[i].miny,sn1);
      fpos =sn1 +p[i].posInFragBuf;

      get_sn_from_xy(point_x, point_y, yres,0, sn2);
      pic[sn2].r +=fbuf[fpos].deltaR;
      pic[sn2].g +=fbuf[fpos].deltaG;
      pic[sn2].b +=fbuf[fpos].deltaB;
      }
    }
  }

//device render function k_render1
__global__ void k_render1
  (cu_particle_splotch *p,  int startP,  int endP,
  void *buf, bool a_eq_e, float grayabsorb,
  cu_exptable_info d_exp_info)//, int xres, int yres)
  {
  //first get the index m of this thread
  int m, n=endP-startP;
  m =blockIdx.x *blockDim.x + threadIdx.x;
  if (m >=n)//m goes from 0 to n-1
    return;
  m +=startP;

  //make fbuf the right type
  cu_fragment_AeqE        *fbuf;
  cu_fragment_AneqE       *fbuf1;
  if (a_eq_e)
    fbuf =(cu_fragment_AeqE*) buf;
  else
    fbuf1 =(cu_fragment_AneqE*)buf;

  //now do the calc
  const float rfac=1.5;
  const float powtmp = pow(Pi,1./3.);
  const float sigma0=powtmp/sqrt(2*Pi);
  const float bfak=1./(2*sqrt(Pi)*powtmp);

  int x0s=0, y0s=0;
  float r=p[m].r;
  float posx=p[m].x, posy=p[m].y;
  posx-=x0s; posy-=y0s;
  float rfacr=rfac*r;

  cu_color a=p[m].a, e, q;
  if (!a_eq_e)
    {
    e=p[m].e;
    q.r=e.r/(a.r+grayabsorb);
    q.g=e.g/(a.g+grayabsorb);
    q.b=e.b/(a.b+grayabsorb);
    }

  float radsq = rfacr*rfacr;
  float prefac1 = -0.5/(r*r*sigma0*sigma0);
  float prefac2 = -0.5*bfak/p[m].ro;
  int minx, miny, maxx, maxy;
  minx =p[m].minx;    miny =p[m].miny;
  maxx =p[m].maxx;    maxy =p[m].maxy;
  unsigned int    fpos;
  fpos =p[m].posInFragBuf -p[startP].posInFragBuf;

  for (int x=minx; x<maxx; ++x)
    {
    float xsq=(x-posx)*(x-posx);
    for (int y=miny; y<maxy; ++y)
      {
      float dsq = (y-posy)*(y-posy) + xsq;
      if (dsq<radsq)
        {
        float fac = prefac2*get_exp(prefac1*dsq, d_exp_info);
        if (a_eq_e)
          {
          fbuf[fpos].deltaR = (fac*a.r);
          fbuf[fpos].deltaG = (fac*a.g);
          fbuf[fpos].deltaB = (fac*a.b);
          }
        else
          {
          float   exp;
          exp =get_exp(fac*a.r, d_exp_info);
          fbuf1[fpos].factorR =exp;
          fbuf1[fpos].deltaR  =q.r*(1.0-exp);
          exp =get_exp(fac*a.g, d_exp_info);
          fbuf1[fpos].factorG =exp;
          fbuf1[fpos].deltaG  =q.g*(1.0-exp);
          exp =get_exp(fac*a.b, d_exp_info);
          fbuf1[fpos].factorB =exp;
          fbuf1[fpos].deltaB  =q.b*(1.0-exp);
          }//if a_eq_e
        }//if dsq<radsq
      else
        {
        if (a_eq_e)
          {
          fbuf[fpos].deltaR =0.0;
          fbuf[fpos].deltaG =0.0;
          fbuf[fpos].deltaB =0.0;
          }
        else
          {
          fbuf1[fpos].deltaR =0.0;
          fbuf1[fpos].deltaG =0.0;
          fbuf1[fpos].deltaB =0.0;
          fbuf1[fpos].factorR =1.0;
          fbuf1[fpos].factorG =1.0;
          fbuf1[fpos].factorB =1.0;
          }
        }
      //for each (x,y)
      fpos++;
      }//y
    }//x
  }

//colorize by kernel
__global__ void k_colorize
  (cu_param_colorize *params, cu_particle_sim *p, int n, cu_particle_splotch *p2,
  cu_colormap_info info)
  {
  //first get the index m of this thread
  int m=blockIdx.x *blockDim.x + threadIdx.x;
  if (m >n)
    m =n;

  //now do the calc, p[m]--->p2[m]
  p2[m].isValid=false;
  if (p[m].z<=0 || p[m].z<=params->zminval || p[m].z>=params->zmaxval)
    return;

  float r=p[m].r;
  float posx=p[m].x, posy=p[m].y;

  float rfacr=params->rfac*r;

  int minx=int(posx-rfacr+1);
  if (minx>=params->res) return;
  minx=max(minx,0);

  int maxx=int(posx+rfacr+1);
  if (maxx<=0) return;
  maxx=min(maxx,params->res);
  if (minx>=maxx) return;

  int miny=int(posy-rfacr+1);
  if (miny>=params->ycut1) return;
  miny=max(miny,params->ycut0);

  int maxy=int(posy+rfacr+1);
  if (maxy<=params->ycut0) return;
  maxy=min(maxy,params->ycut1);
  if (miny>=maxy) return;

  //set region info to output the p2
  p2[m].minx =minx;  p2[m].miny =miny;
  p2[m].maxx =maxx;  p2[m].maxy =maxy;

  float col1=p[m].C1,col2=p[m].C2,col3=p[m].C3;
  clamp (0.0000001,0.9999999,col1);
  if (params->col_vector[p[m].type])
    {
    clamp (0.0000001,0.9999999,col2);
    clamp (0.0000001,0.9999999,col3);
    }
  float intensity=p[m].I;
  clamp (0.0000001,0.9999999,intensity);
  intensity *= params->brightness[p[m].type];

  cu_color e;
  if (params->col_vector[p[m].type])
    {
    e.r=col1*intensity;
    e.g=col2*intensity;
    e.b=col3*intensity;
    }
  else
    {
    e=get_color(p[m].type, col1, info);
    e.r *=intensity;
    e.g *=intensity;
    e.b *=intensity;
    }

  cu_color a=e;

  p2[m].isValid =true;
  p2[m].x =p[m].x;
  p2[m].y =p[m].y;
  p2[m].r =p[m].r;
  p2[m].ro=p[m].ro;
  p2[m].a=a;
  p2[m].e=e;
  }

//Range by kernel step 1
__global__ void k_range1(cu_param_range *pr, cu_particle_sim *p, int n)
  {
  //first get the index m of this thread
  int m=blockIdx.x *blockDim.x + threadIdx.x;
  if (m >=n)
    m =n;

  //now do the calc
  //I, minint, maxint
  if (pr->log_int[p[m].type]) //could access invalid address under EMULATION
    p[m].I = log10(p[m].I);

  //C1, mincol, maxcol
  if (pr->log_col[p[m].type])
  p[m].C1 = log10(p[m].C1);
  if (pr->asinh_col[p[m].type])
    p[m].C1 = my_asinh(p[m].C1);

  //C2, C3, mincol, maxcol
  if (pr->col_vector[p[m].type])
    {
    if (pr->log_col[p[m].type])
      {
      p[m].C2 = log10(p[m].C2);
      p[m].C3 = log10(p[m].C3);
      }
    if (pr->asinh_col[p[m].type])
      {
      p[m].C2 = my_asinh(p[m].C2);
      p[m].C3 = my_asinh(p[m].C3);
      }
    }
  }

//Range by kernel step 2
__global__ void k_range2
  (cu_param_range *pr, cu_particle_sim *p, int n, int itype,
  float minval_int, float maxval_int,
  float minval_col, float maxval_col)
  {
  //first get the index m of this thread
  int m=blockIdx.x *blockDim.x + threadIdx.x;
  if (m >n)
    m =n;

  //do the calculation
  if(p[m].type == itype)///clamp into (min,max)
    {
    my_normalize(minval_int,maxval_int,p[m].I);
    my_normalize(minval_col,maxval_col,p[m].C1);
    if (pr->col_vector[p[m].type])
      {
      my_normalize(minval_col,maxval_col,p[m].C2);
      my_normalize(minval_col,maxval_col,p[m].C3);
      }
    }
  }

//Transform by kernel
__global__ void k_transform
  (cu_particle_sim *p, int n, cu_param_transform *ptrans)
  {
  //first get the index m of this thread
  int m=blockIdx.x *blockDim.x + threadIdx.x;
  if (m >n)
    m =n;

  //copy parameters to __share__ local memory? later

  //now do x,y,z
  float x,y,z;
  x =p[m].x*ptrans->p[0] + p[m].y*ptrans->p[1] + p[m].z*ptrans->p[2] + ptrans->p[3];
  y =p[m].x*ptrans->p[4] + p[m].y*ptrans->p[5] + p[m].z*ptrans->p[6] + ptrans->p[7];
  z =p[m].x*ptrans->p[8] + p[m].y*ptrans->p[9] + p[m].z*ptrans->p[10]+ ptrans->p[11];
  p[m].x =x;
  p[m].y =y;
  p[m].z =z;

  //do ro and r
  float   xfac =ptrans->xfac;
  if (!ptrans->projection)
    {
    p[m].x = ptrans->res*.5 * (p[m].x+ptrans->fovfct*ptrans->dist)*xfac;
    p[m].y = ptrans->res*.5 * (p[m].y+ptrans->fovfct*ptrans->dist)*xfac;
    }
    else
    {
    xfac=1./(ptrans->fovfct*p[m].z);
    p[m].x = ptrans->res*.5 * (p[m].x+ptrans->fovfct*p[m].z)*xfac;
    p[m].y = ptrans->res*.5 * (p[m].y+ptrans->fovfct*p[m].z)*xfac;
    }

  p[m].ro = p[m].r;
  p[m].r = p[m].r *ptrans->res*.5*xfac;

  if (ptrans->minhsmlpixel)
    if ((p[m].r <= 0.5) && (p[m].r >= 0.0))
      {
      p[m].r = 0.5;
      p[m].ro = p[m].r/(ptrans->res*.5*xfac);
      }
  }

#endif // #ifndef SPLOTCH_KERNEL_H
