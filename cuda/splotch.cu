#include "hip/hip_runtime.h"
/*
Try accelating splotch with CUDA. July 2009. 
*/


// includes, system
#include <stdlib.h>
#include <stdio.h>
//#include <string.h>
#include <math.h>

// includes, project
#include "hip/hip_runtime.h"
#include <cutil_inline.h>

// includes, kernels
#include <splotch_kernel.cu> 
#include "vtimer.h"
#include "splotch_cuda.h"
#include "CuPolicy.h"


//functions defs
void dump_pr(cu_param_range *pr);

//////////////////////////////
//global varibles
float       *d_tmp=0;   //used for debug
float       *d_expTable =0;
CuPolicy    *policy=0;
cu_particle_sim  *d_pd=0; //device_particle_data
//////////////////////////////

extern "C" 
void    cu_init()
{
    //initilize cuda runtime
    hipSetDevice( cutGetMaxGflopsDeviceId() );

    unsigned int s;

    //d_tmp used for debug
    s =sizeof (float);
    cutilSafeCall( hipMalloc((void**) &d_tmp, s));
    //copy 0.0 to *d_tmp;
    float   f=0.0;
    cutilSafeCall(hipMemcpy(d_tmp, &f, sizeof(float),
                              hipMemcpyHostToDevice) );    

    //Initialize policy class
    policy =new CuPolicy();
}

extern "C"
void	cu_end()
{
    // clean up memory
    cutilSafeCall(hipFree(d_tmp));
    cutilSafeCall(hipFree(d_pd));
    hipDeviceReset();

    //clear policy object
    if (policy)
        delete policy;
}

extern "C"
void	cu_range(paramfile &params ,cu_particle_sim* h_pd, unsigned int n)
{
    //allocate device memory for particle data
    int s =policy->GetSizeDPD(n);
#ifdef _DEVICEEMU
    printf("device_particle_data size:%d" ,s);
#endif
    //one more space allocated for the dumb
    cutilSafeCall( hipMalloc((void**) &d_pd, s +sizeof(cu_particle_sim)));
    
    //copy particle data to device
    cutilSafeCall(hipMemcpy(d_pd, h_pd, s,
                              hipMemcpyHostToDevice) );    
    //ask for dims from policy
    dim3    dimGrid, dimBlock;
    policy->GetDimsRange(&dimGrid, &dimBlock);    

    //prepare parameters for stage 1
    cu_param_range  pr;
    int ptypes = params.find<int>("ptypes",1);
    pr.ptypes =ptypes;
    //now collect parameters from configuration
    for(int itype=0;itype<ptypes;itype++)
    {
        pr.log_int[itype] = params.find<bool>("intensity_log"+dataToString(itype),true);
        pr.log_col[itype] = params.find<bool>("color_log"+dataToString(itype),true);
        pr.asinh_col[itype] = params.find<bool>("color_asinh"+dataToString(itype),false);
        pr.col_vector[itype] = params.find<bool>("color_is_vector"+dataToString(itype),false);
        pr.mincol[itype]=1e30;
        pr.maxcol[itype]=-1e30;
        pr.minint[itype]=1e30;
        pr.maxint[itype]=-1e30;
    }
    //allocate memory on device and dump parameters to it
    cu_param_range  *d_pr=0;
    s =sizeof(cu_param_range);
    cutilSafeCall( hipMalloc((void**) &d_pr, s) );    
    cutilSafeCall(hipMemcpy(d_pr, &pr, s,
                              hipMemcpyHostToDevice) );    

    // call device for stage 1
    k_range1<<<dimGrid,dimBlock>>>(d_pr, d_pd, n);
    
    // copy out pr which were changed, for stage 2
    s =sizeof(cu_param_range);
    cutilSafeCall(hipMemcpy( &pr, d_pr,  s,
                              hipMemcpyDeviceToHost) );    
dump_pr(&pr);

    // call device for stage 2 ptypes times
    // prepare parameters1 first
    for(int itype=0;itype<ptypes;itype++)
    {
        float minval_int = params.find<float>("intensity_	min"+dataToString(itype),pr.minint[itype]);
        float maxval_int = params.find<float>("intensity_max"+dataToString(itype),pr.maxint[itype]);
        float minval_col = params.find<float>("color_min"+dataToString(itype),pr.mincol[itype]);
        float maxval_col = params.find<float>("color_max"+dataToString(itype),pr.maxcol[itype]);
        
        k_range2<<<dimGrid, dimBlock>>>(d_pd, n, minval_int,maxval_int,minval_col,maxval_col);
    }

    //copy result out to host
    cutilSafeCall(hipMemcpy(h_pd, d_pd, s,
                              hipMemcpyDeviceToHost) );    
    
    //free parameters on device
     cutilSafeCall(hipFree(d_pr));

    //d_pd will be freed in cu_end
}

void dump_pr(cu_param_range *pr)
{
    printf("\ndump_pr:\n");
    printf("col_vector, log_int,log_col,asinh_col,");
    printf("mincol, maxcol, minint,maxint\n");

    for (int i=0; i<pr->ptypes; i++)
    {
        printf("%d, %d, %d, %d, %f, %f, %f, %f\n",
            pr->col_vector[i], pr->log_int[i], 
            pr->log_col[i],	pr->asinh_col[i], pr->mincol[i],
    		pr->maxcol[i], pr->minint[i], pr->maxint[i]);
    }                     
}

/*
extern "C" 
void    cu_initExp(int nExp, float *h_expTable)
{
    //exp table 
    int    s= sizeof(float) *nExp;
    cutilSafeCall( hipMalloc((void**) &d_expTable, s));        
    cutilSafeCall(hipMemcpy(d_expTable, h_expTable, s,
                              hipMemcpyHostToDevice) );    

}

extern "C" 
void    cu_initGVars(G_VARS *h_vars)
{
    //g_vars
    int    s =sizeof(G_VARS);
    cutilSafeCall( hipMalloc((void**) &d_g_vars, s));
    cutilSafeCall(hipMemcpy(d_g_vars, h_vars, s,
                              hipMemcpyHostToDevice) );    

}

extern "C" 
void    cu_initPArray(PARTICLE *h_p)
{
    //particle array
    int    s =sizeof(PARTICLE) *nParticle;
    cutilSafeCall( hipMalloc((void**) &d_p, s));
    cutilSafeCall(hipMemcpy(d_p, h_p, s,
                              hipMemcpyHostToDevice) );    

}

extern "C" 
void    cu_copyPArray2host(PARTICLE *h_p)
{
    //particle array
    int    s =sizeof(PARTICLE) *nParticle;
    cutilSafeCall(hipMemcpy( h_p, d_p, s,
                              hipMemcpyDeviceToHost) );    
}

extern "C" 
void    cu_preCalc()
{
    k_preCalc<<<512,512>>>( d_g_vars, d_p);
}

extern "C" 
void    cu_init_frameBuf()
{
    int    s =sizeof(FRAGMENT) *800*800 *64;//is it ok for 64 frame buffers? Yes.
    cutilSafeCall( hipMalloc((void**) &d_f, s));
    
    dim3 dimGrid(800,800);
    k_initFBuf<<<dimGrid,64>>>(d_f);
}

extern "C"
void    cu_copyFrameBuf2host(FRAGMENT   *h_f)
{
    int    s =sizeof(FRAGMENT) *800*800;
    cutilSafeCall(hipMemcpy( h_f, d_f, s,
                              hipMemcpyDeviceToHost) );            
//    int error=hipMemcpy( h_f, d_f+63*800*800, s,//......
//                hipMemcpyDeviceToHost);
//    printf("\nerrorCode=%d",error);
}

extern "C"
void    cu_shadeA(int nStart, int nEnd) //do shading due to Plan A
{
    k_shadeA<<<64,1>>> (nStart, nEnd, d_p, d_f, d_g_vars,d_expTable);
}

extern "C"
void	cu_end()
{
    // clean up memory
    cutilSafeCall(hipFree(d_tmp));
    cutilSafeCall(hipFree(d_expTable));
    cutilSafeCall(hipFree(d_g_vars));
    cutilSafeCall(hipFree(d_p));
    cutilSafeCall(hipFree(d_f));
    
    hipDeviceReset();
}

extern "C"
void	cu_combineA()
{
    dim3    dimGrid(800,800);//dimGrid(40,40); with 400
    k_combineA<<<dimGrid, 1>>>(d_f);
}
*/

#ifdef CU_DO_TESTS
/*
extern "C"
void	cu_check1()
{
    //check if d_g_var is correctly assigned
    G_VARS  gv;
    cutilSafeCall(hipMemcpy(&gv, d_g_vars, sizeof(gv),
                              hipMemcpyDeviceToHost) );
    printf("\ncu_check1: %f,%f,%f,%f,%f,%f,%d,%d,%d",
        gv.rfac, gv.bfak, gv.i00, gv.sigma0, gv.brightness,
        gv.grayabsorb, gv.res, gv.ycut0, gv.ycut1);

    //check if d_p is correctly assigned for computing
    PARTICLE    *p =new PARTICLE[nParticle];
    cutilSafeCall(hipMemcpy(p, d_p, sizeof(PARTICLE)*nParticle,
                              hipMemcpyDeviceToHost) );
//    not allowed! cutilSafeCall(hipMemcpy(&p2, d_p+ (nParticle-1)*sizeof(p1), sizeof(p1),
  //                            hipMemcpyDeviceToHost) );
//    cutilSafeCall(hipMemcpy(&p3, d_p+ (nParticle-1)/2*sizeof(p1),sizeof(p1),
//                              hipMemcpyDeviceToHost) );
    printf("\ncu_check1: %f,%f,%f,%f,%f,%f,%f,%d",
        p[0].x, p[0].y, p[0].z, p[0].r, p[0].ro, p[0].I, p[0].T, p[0].type);
    printf("\ncu_check1: %f,%f,%f,%f,%f,%f,%f,%d",
        p[8888].x, p[8888].y, p[8888].z, p[8888].r, p[8888].ro, p[8888].I, p[8888].T, p[8888].type);
    printf("\ncu_check1: %f,%f,%f,%f,%f,%f,%f,%d",
        p[nParticle-1].x, p[nParticle-1].y, p[nParticle-1].z, p[nParticle-1].r, p[nParticle-1].ro, p[nParticle-1].I, p[nParticle-1].T, p[nParticle-1].type);
    delete p;
    
}


extern "C"
float   cu_testExp(float arg)
{
    float   result;
    //call kernel1 to do xexp( arg);
    k_xexp<<<1,1>>> (10000,-20., arg, d_tmp, d_expTable);
    //copy the result on device back to host
    unsigned int    s =sizeof (float);
//    cutilSafeCall(hipMemcpy(&result, d_tmp, s,
//                              hipMemcpyDeviceToHost) );    //highly time consuming
    
    //return    it to caller
    return result;
}

extern "C"
double cu_TestDouble()
{
    float  *d_result, result;
    int s =sizeof(result);
    
    cutilSafeCall( hipMalloc((void**) &d_result, s));
    k_testDouble<<<1,1>>> (d_result);
    cutilSafeCall(hipMemcpy(&result, d_result, s,
                              hipMemcpyDeviceToHost) );
    cutilSafeCall(hipFree(d_result));
    return result;    
}



extern "C"
float cu_test1(int n, float  *table, float arg)
{
    float result =1;

    //allocate space on device for exp table 
    unsigned int s =sizeof(float) *n;
    float   *d_expTable =0;
    cutilSafeCall( hipMalloc((void**) &d_expTable, s));        

    //copy exp table from host to device
    cutilSafeCall(hipMemcpy(d_expTable, table, s,
                              hipMemcpyHostToDevice) );    
    

    //verify things
    //allcate a host mem for copying device data out
    float *h_temp =(float*)malloc(s);
    //copy device exp table to this temp memory, from device to host
    cutilSafeCall(hipMemcpy(h_temp, d_expTable, s,
                              hipMemcpyDeviceToHost) );    
    //compare the two tables
    for (int i=0; i<n; i++)
    {
        if ( h_temp[i] != table[i] )
        {
            result =-1;
            break;
        }
        if ( i==0 || i==n/2 || i==n-1) //pick some to check
            printf("\n %f == %f", h_temp[i], table[i]);
    }

    //test calling a kernel function to retrieve a table element
    //allocate the memory for the single float result on device
    float   *d_tmp;
    s =sizeof (float);
    cutilSafeCall( hipMalloc((void**) &d_tmp, s));

    //call kernel to get d_expTable[10] to d_tmp
    test_kernel <<<1,1>>>( d_expTable, 10, d_tmp);       
    //copy the result on device back to host
    float   a;
    cutilSafeCall(hipMemcpy(&a, d_tmp, s,
                              hipMemcpyDeviceToHost) );    
    //and compare
    if ( table[10] != a)
        result =-1;

    //call kernel1 to do xexp( arg);
    k_xexp<<<1,1>>> (10000,-20., arg, d_tmp, d_expTable);
    //copy the result on device back to host
    cutilSafeCall(hipMemcpy(&a, d_tmp, s,
                              hipMemcpyDeviceToHost) );    
    //return    it to caller
    result =a;

    //free the memory on host
    free(h_temp);

    //free the device memory
    cutilSafeCall(hipFree(d_expTable));
    cutilSafeCall(hipFree(d_tmp));    

    return result;
}


extern "C"
void cu_test2(int *l)
{
    VTimer t;

    int    *d_l;
    int sz =sizeof(int) *65535*512;
    

    cutilSafeCall( hipMalloc((void**) &d_l, sz));       
    
    t.start();
    test_kernel2<<<65535,512>>>(d_l);
	t.stop();
	printf("\ntime1=%f", t.getTime());
    
    int err= hipMemcpy(l, d_l, sz, hipMemcpyDeviceToHost) ;    

    hipFree(d_l);  
}
*/
#endif //CU_DO_TESTS